#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
namespace Efficient {

__global__ void gpu_scan_up(int n, int d, int *odata, int *idata) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	if( index<n ) {
		if( index>0 && (index+1)%(1<<d)==0 ) {
			odata[index] = idata[index] + idata[index - (1<<(d-1)) ];
		} else {
			odata[index] = idata[index];
		}
//		if( index%( 1<<(d+1) ) ==0 ) {
//			odata[index + (1<<(d+1)) - 1] = idata[index + (1<<(d+1)) - 1] + idata[ index + (1<<d) - 1 ];
//		} else {
//			odata[index] = idata[index];
//		}
	}
}

__global__ void gpu_scan_down(int n, int d, int *odata, int *idata) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	if( index<n ) {
		if( index%( 1<<(d+1) ) ==0 ) {
			int t = idata[index + (1<<d) - 1];
			odata[index + (1<<d) - 1] = idata[index + (1<<(d+1)) - 1];
			odata[index + (1<<(d+1)) - 1] = idata[index + (1<<(d+1)) - 1] + t;
		} else {
			odata[index] = idata[index];
		}
	}
}

/**
 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
 */
void scan(int n, int *odata, const int *idata) {
	int *dev_idata;
	int *dev_odata;
	int blockSize = 1024;

    int n_new = 1 << ilog2ceil(n);
    //printf("n_new is %d \n", n_new);
	int idata_new[n_new];
    for(int i=0; i<n_new; i++) {
    	if(i<n) {
    		idata_new[i] = idata[i];
    	} else {
    		idata_new[i] = 0;
    	}
    }
    dim3 fullBlocksPerGrid((n_new + blockSize - 1) / blockSize);

    hipMalloc((void**)&dev_idata, n_new * sizeof(int));
    hipMalloc((void**)&dev_odata, n_new * sizeof(int));

    hipMemcpy(dev_idata, idata_new, n_new * sizeof(int), hipMemcpyHostToDevice);

	float time = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
    for(int j=1; j<=ilog2ceil(n); j++) {
    	gpu_scan_up<<<fullBlocksPerGrid, blockSize>>>(n_new, j, dev_odata, dev_idata);
    	//hipMemcpy(dev_idata, dev_odata, n_new * sizeof(int), hipMemcpyDeviceToDevice);
    	dev_idata = dev_odata;
    }
    hipMemcpy(idata_new, dev_odata, n_new * sizeof(int), hipMemcpyDeviceToHost);

    idata_new[n_new-1] = 0;
    hipMemcpy(dev_idata, idata_new, n_new * sizeof(int), hipMemcpyHostToDevice);
    for(int k=ilog2ceil(n)-1; k>=0; k--) {
    	gpu_scan_down<<<fullBlocksPerGrid, blockSize>>>(n_new, k, dev_odata, dev_idata);
    	//hipMemcpy(dev_idata, dev_odata, n_new * sizeof(int), hipMemcpyDeviceToDevice);
    	dev_idata = dev_odata;
    }
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("GPU work-efficient scan time is %.4f ms \n", time);

    hipMemcpy(odata, dev_odata, n_new * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dev_idata);
    hipFree(dev_odata);
}

/**
 * Performs stream compaction on idata, storing the result into odata.
 * All zeroes are discarded.
 *
 * @param n      The number of elements in idata.
 * @param odata  The array into which to store elements.
 * @param idata  The array of elements to compact.
 * @returns      The number of elements remaining after compaction.
 */
int compact(int n, int *odata, const int *idata) {

	int *dev_idata;
	int *dev_odata;
	int *dev_bools;
	int *dev_indices;

	int hst_bools[n];
	int hst_indices[n];

	int blockSize = 1024;

	dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

    hipMalloc((void**)&dev_idata, n * sizeof(int));
    hipMalloc((void**)&dev_odata, n * sizeof(int));
    hipMalloc((void**)&dev_bools, n * sizeof(int));
    hipMalloc((void**)&dev_indices, n * sizeof(int));

    hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
    Common::kernMapToBoolean<<<fullBlocksPerGrid, blockSize>>>(n, dev_bools, dev_idata);
    hipMemcpy(hst_bools, dev_bools, n * sizeof(int), hipMemcpyDeviceToHost);

	scan(n, hst_indices, hst_bools);
	//printf("n is %d \n", n);

	hipMemcpy(dev_indices, hst_indices, n * sizeof(int), hipMemcpyHostToDevice);
	Common::kernScatter<<<fullBlocksPerGrid, blockSize>>>(n, dev_odata, dev_idata, dev_bools, dev_indices);
	hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_idata);
	hipFree(dev_odata);
	hipFree(dev_bools);
	hipFree(dev_indices);

	if(idata[n-1] == 0) {
		return hst_indices[n-1];
	} else {
		return hst_indices[n-1] + 1;
	}

}

}
}
